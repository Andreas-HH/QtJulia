#include "hip/hip_runtime.h"
#include <JuliaMath.h>

__device__ void multiplyMyC_gpu(myComplex x, myComplex a) {
  double xReal = x.real;
  double xImg  = x.img;
  double aReal = a.real;
  double aImg  = a.img;
  
  x.real = xReal*aReal - xImg*aImg;
  x.img  = xImg*aReal + xReal*aImg;
}

__device__ void addMyC_gpu(myComplex x, myComplex a) {
  x.real = x.real + a.real;
  x.img  = x.img + a.img;
}

__device__ double norm2_gpu(myComplex x) { 
  return x.real * x.real + x.img * x.img; 
}

__global__ void juliaKernel(myComplex *coords, double *data, myComplex *c, int threshold, int width, int height) {
  int i;
  int idx, cacheIdx;
  int x, y;
  __shared__ myComplex xArray[256];
  myComplex lC = *c;
  
  x = 16*blockIdx.x + threadIdx.x;
  y = 16*blockIdx.y + threadIdx.y;
//   idx = 16*gridDim.x*y + x;
  idx = y*width + x;
  cacheIdx = 16*threadIdx.y + threadIdx.x;
  
  xArray[cacheIdx] = coords[idx];
  
  data[idx] = 1.;
  if (x < width && y < height) {// && idx < width*height) {
    for (i = 0; i < threshold; i++) {
      multiplyMyC_gpu(xArray[cacheIdx], lC);
      addMyC_gpu(xArray[cacheIdx], lC);
    
      if (norm2_gpu(xArray[cacheIdx]) > 1000.) {
        break;
      }
    }
//     data[idx] = ((double) i)/((double) threshold);
    data[idx] = 1.;
  }
}

__global__ void initCoords(myComplex *coords, double topLr, double topLi, double pixelW, int width, int height) {
  int idx;
  int x, y;
  
  x = 16*blockIdx.x + threadIdx.x;
  y = 16*blockIdx.y + threadIdx.y;
  idx = y*width + x;
//   idx = 16*gridDim.x*y + x;
//   idx = 16*blockIdx.x + 16*gridDim.x*blockIdx.y + threadIdx.x + 16*threadIdx.y;
//   x = idx % gridDim.x*blockDim.x;
//   y = idx / gridDim.x*blockDim.x;
  if (x < width && y < height) {// && idx < width*height) {
    coords[idx].real = topLr + x*pixelW;
    coords[idx].img = topLi - y*pixelW;
  }
}

extern "C" {
  void juliaValueGPU(myComplex *coords, double *data, myComplex *c, int width, int height, int threshold) {
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, data);
    printf("data on device: %i\n", attr.device);
    
    dim3 grid((width+15)/16, (height+15)/16);
    dim3 block(16, 16);
    juliaKernel<<<grid,block>>>(coords, data, c, threshold, width, height);
    printf("Der kernel sollte eig fertig sein...\n");
  }
  
  void downloadData(double *data_gpu, double *data_cpu, int N) {
    printf("downloading data...\n");
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, data_gpu);
    if (attr.memoryType == hipMemoryTypeHost) {
      printf("error: gpu\n");
    }
    hipPointerGetAttributes(&attr, data_cpu);
    if (attr.memoryType == hipMemoryTypeDevice) {
      printf("error: cpu\n");
    }
    
    hipMemcpy(data_cpu, data_gpu, N*sizeof(double), hipMemcpyDefault);
  }
  
  void uploadC(myComplex *c_gpu, myComplex *c_cpu) {
    printf("uploadC()\n");
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, c_cpu);
    if (attr.memoryType == hipMemoryTypeDevice) {
      printf("error: cpu\n");
    }
    hipPointerGetAttributes(&attr, c_gpu);
    if (attr.memoryType == hipMemoryTypeHost) {
      printf("error: gpu\n");
    }

//     myComplex *test = createMyComplex(0., 0.);
    hipMemcpy(c_gpu, c_cpu, sizeof(myComplex), hipMemcpyDefault);
//     hipMemcpy(test, c_gpu, sizeof(myComplex), hipMemcpyDefault);
//     printf("c_gpu = (%f. %f)\n", test->real, test->img);
  }
  
  /*myComplex* createEmptyMyComplexOnGPU() {
    myComplex *a_gpu;
    
    hipMalloc(&a_gpu, sizeof(myComplex));
  
    return a_gpu;
  }*/
  
  myComplex* createMyComplexOnGPU(myComplex *value_host) {
    printf("createMyComplexOnGPU()\n");
//     hipSetDevice(1);
    myComplex *a_gpu;
    hipError_t error;
    hipPointerAttribute_t attr;
    
    hipHostRegister(value_host, sizeof(myComplex), hipHostRegisterMapped);
    
    error = hipMalloc(&a_gpu, sizeof(myComplex));
    
    
    hipPointerGetAttributes(&attr, value_host);
    if (attr.memoryType == hipMemoryTypeDevice) {
      printf("error: cpu\n");
    }
    hipPointerGetAttributes(&attr, a_gpu);
    if (attr.memoryType == hipMemoryTypeHost) {
      printf("error: gpu\n");
    }
    
    hipMemcpy(a_gpu, value_host, sizeof(myComplex), hipMemcpyHostToDevice);
    
    hipPointerGetAttributes(&attr, value_host);
    if (attr.memoryType == hipMemoryTypeDevice) {
      printf("now error: cpu\n");
    }
    hipPointerGetAttributes(&attr, a_gpu);
    if (attr.memoryType == hipMemoryTypeHost) {
      printf("now error: gpu\n");
    }
    
    
    if (error == hipSuccess) {
      printf("successfully uploaded a myComplex\n");
    }
    
//     hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, a_gpu);
    printf("a_gpu on device: %i\n", attr.device);
    
    hipHostUnregister(value_host);
  
    return a_gpu;
  }
  
  double* initDataGPU(int width, int height) {
    double *data_gpu;
    hipError_t error;
    
    error = hipMalloc(&data_gpu, width*height*sizeof(double));
    if (error == hipSuccess) {
      printf("GPU data allocated!\n");
    }
    
    hipPointerAttribute_t attr;
    hipPointerGetAttributes(&attr, data_gpu);
    printf("data_gpu on device: %i\n", attr.device);
    
    return data_gpu;
  }
  
  // fill using a kernel !! (Erlaubt auch die alte Struktur mit myComplex**)
  myComplex* initCoordsGPU(int width, int height, myComplex* screenMid_host, double pixelW) {
//     int x, y;
    myComplex *coords;
    hipError_t error;
    double topLeftX = screenMid_host->real - (width/2.)*pixelW;
    double topLeftY = screenMid_host->img + (height/2.)*pixelW;
//     myComplex *current_point = createMyComplex(topLeftX, topLeftY);
//     myComplex *tmp_point = createMyComplex(1., 1.);
    
    error = hipMalloc(&coords, width*height*sizeof(myComplex));
    if (error == hipSuccess) {
      printf("hipMalloc ausgefuehrt\n");
    }
    dim3 grid((width+15)/16, (height+15)/16);
    dim3 block(16, 16);
    initCoords<<<grid, block>>>(coords, topLeftX, topLeftY, pixelW, width, height);
    
    myComplex *test = createMyComplex(topLeftX, topLeftY);
    hipMemcpy(test, coords+1610, sizeof(myComplex), hipMemcpyDefault);
    printf("von GPU coords gelesen: (%f, %f), pW = %f\n", test->real, test->img, pixelW);
//     printf("kernel fedd�sch\n");
    /*for (x = 0; x < width; x++) {
      for (y = 0; y < height; y++) {
// 	hipMalloc(coords + y*width + x, sizeof(myComplex));
	hipMemcpy(coords + y*width + x, current_point, sizeof(myComplex), hipMemcpyDefault);
        current_point->img -= pixelW;
      }
      current_point->img = topLeftY;
      current_point->real += pixelW;
    }
    free(current_point);*/
    
    return coords;
  }
  
  void clearData(double *data) {
    hipFree(data);
  }
  
  void clearCoords(myComplex *coords) {
//     int i;
    
    /*for (i = 0; i < width*height; i++) {
      hipFree(coords + i);
    }*/
    hipFree(coords);
  }
}